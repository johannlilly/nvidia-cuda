#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2018 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include "ShaderStructs.h"

__global__ void sinewave_gen_kernel(Vertex *vertices, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float) width;
    float v = y / (float) height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

    if (y < height && x < width)
    {
        // write output vertex
        vertices[y*width+x].position.x = u;
        vertices[y*width+x].position.y = w;
        vertices[y*width+x].position.z = v;
        //vertices[y*width+x].position[3] = 1.0f;
        vertices[y*width+x].color.x = 1.0f;
        vertices[y*width+x].color.y = 0.0f;
        vertices[y*width+x].color.z = 0.0f;
		vertices[y*width + x].color.w = 0.0f;
    }
}

// The host CPU Sinewave thread spawner
void RunSineWaveKernel(unsigned int mesh_width, unsigned int mesh_height, Vertex *cudaDevVertptr, hipStream_t streamToRun, float AnimTime)
{
	dim3 block(16, 16, 1);
	dim3 grid(mesh_width / 16, mesh_height / 16, 1);
	Vertex *vertices = (Vertex*)cudaDevVertptr;
	sinewave_gen_kernel<<< grid, block, 0, streamToRun >>>(vertices, mesh_width, mesh_height, AnimTime);

	getLastCudaError("sinewave_gen_kernel execution failed.\n");
}

